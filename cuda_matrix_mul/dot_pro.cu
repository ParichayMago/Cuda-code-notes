#include "hip/hip_runtime.h"
#include <stdio.h>

int main() {
  int n = 5;
  int size = n * sizeof(int);

  int a[] = {1,2,3,4,5};
  int b[] = {1,2,3,4,5};
  int c[] = {0};

  int *d_a, *d_b, *d_c;

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

  vector_add<<<1, n>>>(d_a, d_b, d_c, n);



  return 0
}


__global__ void vector_add(int a**, int b**, int **c; int size) {
  for (int i = 0; i <size; i++) {
    c[i] = a[i] * b[i];
  }
  vector_sum();

}


__global__ void vector_sum(int a**, int size){
  int sum =0;
  for(int i =0; i<size; i++) {
    sum += a[i];
  }
  return;
}