
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to add elements of two arrays
__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int n = 5;
    int a[5] = {1, 2, 3, 4, 5};
    int b[5] = {10, 20, 30, 40, 50};
    int c[5] = {0};

    int *d_a, *d_b, *d_c;
    int size = n * sizeof(int);

    // Allocate memory on the device
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<1, n>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("Result:\n");
    for (int i = 0; i < n; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Free memory on the device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
