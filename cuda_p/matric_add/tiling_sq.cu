#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

// Assuming equal size sq matrix..

int main()
{
  int a[4][4] = {{1, 2, 3, 4}, {1, 2, 3, 4}, {1, 2, 3, 4}, {1, 2, 3, 4}};
  int b[4][4] = {{1, 2, 3, 4}, {1, 2, 3, 4}, {1, 2, 3, 4}, {1, 2, 3, 4}};
}

// Splitting data into smaller tiles.
// Load tiles into shared memory.
// performing dot product with the shared memory.

__global__ void tiled_sq_matrix_mul_kernel(float *A, float *B, float *C, int N)
{
  // tile width NxN factors down to N/w*t
  int TILE_WIDTH = 2;

  // details regarding the thread
  int by = blockIdx.y;
  int bx = blockIdx.x;

  int ty = threadIdx.y;
  int tx = threadIdx.x;

  // working on C[i,j]
  int Rows = by * TILE_WIDTH + ty;
  int Cols = bx * TILE_WIDTH + tx;

  __shared__ float As[TILE_WIDTH][TILE_WIDTH];
  __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

  int p_value = 0;
  for (int phase = 0; phase < N / TILE_WIDTH; phase++)
  {
    As[tx][ty] = A[Rows * N + phase * TILE_WIDTH + tx];
    Bs[tx][ty] = B[(phase * TILE_WIDTH + ty) * N + Cols];
    __syncthreads();

    for (int k = 0; k < TILE_WIDTH; k++)
    {
      p_value += As[ty][k] * Bs[k][tx];
    }
    __syncthreads();
  }
  C[Rows * N + Cols] = p_value;
}
