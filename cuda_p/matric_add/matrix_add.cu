#include <stdio.h>
#include <hip/hip_runtime.h>

int sum(int *a, int size);
__global__ void dot_product(int *a, int *b, int *c, int size);

int main()
{
  int n = 5;
  int size = n * sizeof(int);

  int a[] = {1, 2, 3, 4, 5};
  int b[] = {1, 2, 3, 4, 5};
  int c[5] = {0};

  int *d_a, *d_b, *d_c;

  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  dot_product<<<1, n>>>(d_a, d_b, d_c, n);

  // Ensure the kernel finishes executing
  hipDeviceSynchronize();

  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  int result = sum(c, n);

  printf("Sum of dot product result: %d\n", result);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}

__global__ void dot_product(int *a, int *b, int *c, int size)
{
  int indx = threadIdx.x + blockIdx.x * blockDim.x;

  if (indx < size)
  {
    c[indx] = a[indx] * b[indx];
    printf("c[%d] = %d\n", indx, c[indx]); // Corrected printf
  }
}

int sum(int *a, int size)
{
  int sum = 0;
  for (int i = 0; i < size; i++)
  {
    sum += a[i];
  }

  return sum;
}
