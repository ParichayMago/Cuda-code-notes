#include <stdio.h>
#include <hip/hip_runtime.h>

int main () {
  int a[4][4] = {{1,2,3,4}, {1,2,3,4}, {1,2,3,4}, {1,2,3,4}};
  int b[4][4] = {{1,2,3,4}, {1,2,3,4}, {1,2,3,4}, {1,2,3,4}};

}

// Splitting data into smaller tiles.
// Load tiles into shared memory.
// performing dot product with the shared memory.

__global__ void tiled_sq_matrix_mul_kernel(float* A, float* B, float* C, int N) {
  // tile width NxN factors down to N/w*t
  int TILE_WIDTH = 2;

  // details regarding the thread
  int by = blockIdx.y;
  int bx = blockIdx.x;

  int ty = threadIdx.y;
  int tx = threadIdx.x;

  // working on C[i,j]
  int i = by * blockDim.y + ty;
  int j = bx * blockDim.x + tx;
  

  __shared__ float sh_A[2][2];
  __shared__ float sh_B[2][2];

  int PValue = 0;

  for(int phase=0; phase < N/TILE_WIDTH; phase++) {
    sh_A[ty][tx] = A[(i)*N + phase*TILE_WIDTH + tx];
    sh_B[ty][tx] = B[(phase*TILE_WIDTH + ty) * N+j];
    __syncthreads();

    for(int k = 0; k<TILE_WIDTH; k++) {
      PValue += sh_A[ty][k] * sh_B[k][tx];
      __syncthreads();
    }
    C[i*N+j] = PValue;

  }
}