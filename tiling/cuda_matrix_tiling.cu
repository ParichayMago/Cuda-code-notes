// Steps to perform tiling matrix multiplication
// step 1: Splitting data into smaller tilles
// step 2: Transferring data from G. Mem to shared mem so all the threads in the same block can access it.
// step 3: Dot Product of tiles and storing the value in the registor memroy

#include <hip/hip_runtime.h>
#define ts 2


__global__ void mat_mul_tiled(float* A, float* B, float* C, int N) {
  int by = blockIdx.y;
  int bx = blockIdx.x;

  int ty = threadIdx.y;
  int tx = threadIdx.x;

  int row = ts * by + ty; //i
  int col = ts * bx + tx; //j

  __shared__ float sh_a[ts][ts];
  __shared__ float sh_b[ts][ts];

  float Pvalue = 0;

  for (int phase=0; phase<N/ts ; phase++) {
  // local index is 2x2 as same as block same as tile
  // row is the same as gloabl index of the thread u got this, and column is the phase*N+tx
    sh_a[ty][tx] = A[(row)*N + phase*ts+tx];
    sh_b[ty][tx] = B[(phase*ts + ty) * ts + col];
    __syncthreads();

    for(int k=0; k<ts; k++){
      Pvalue += sh_a[ty][k] * sh_b[k][tx];
    __syncthreads();
    }
    C[row*N + col ] = Pvalue;
  }
}
 