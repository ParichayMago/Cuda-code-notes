#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


int main() {

  int d_K = 6;

  int Q[6] = {0.12, 0.30, 021, 0.12, 0.30, 021};
  int K[6] = {0.12, 0.30, 021, 0.12, 0.30, 021};




  return 0;
}


__global__ void *dotProduct(int* q, int* k ) {
  int thread = threadIdx.x + (blockDim.x * blockIdx.x);
  



}

int* softmax(int *S) {
  int len = sizeof(S) / sizeof(S[1]);
  int exp_sum = 0;
  int P[len];
  for(int i = 0; i<len; i++) {
    exp_sum += exp(S[i]);
  }
  for(int i =0; i<len; i++) {
    P[i] = exp(S[i])/exp_sum;
  }

  return P;

}


